#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#ifndef __HIPCC__  
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h> 
#include ""


#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Cuda error: %s\n", hipGetErrorString(err));    \
printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
}       

const long N = 33554432; 


__global__ void calculationPiGPU(float *x, float *y, int *blocksCounts) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x; 

	int bias = gridDim.x * blockDim.x;

	
	__shared__ int sharedCounts[512]; 

	int countPointsInCircle = 0;
	for (int i = idx; i < N; i += bias) {
		if (x[i] * x[i] + y[i] * y[i] < 1) {
			countPointsInCircle++;
		}
	}
	sharedCounts[threadIdx.x] = countPointsInCircle;

	__syncthreads();

	
	if (threadIdx.x == 0) {
		int total = 0;
		for (int j = 0; j < 512; j++) {
			total += sharedCounts[j];
		}
		blocksCounts[blockIdx.x] = total;
	}
}


float calculationPiCPU(float *x, float *y) {
	int countPointsInCircle = 0; 
	for (int i = 0; i < N; i++) {
		if (x[i] * x[i] + y[i] * y[i] < 1) {
			countPointsInCircle++;
		}
	}
	return float(countPointsInCircle) * 4 / N;
}



int main()
{
	setlocale(LC_ALL, "RUS");
	float *X, *Y, *devX, *devY;

	
	X = (float *)calloc(N, sizeof(float));
	Y = (float *)calloc(N, sizeof(float));

	
	CUDA_CHECK_ERROR(hipMalloc((void **)&devX, N * sizeof(float)));
	CUDA_CHECK_ERROR(hipMalloc((void **)&devY, N * sizeof(float)));

	hiprandGenerator_t curandGenerator; 
	hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_MTGP32); 
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 1234ULL); 
	hiprandGenerateUniform(curandGenerator, devX, N); 
	hiprandGenerateUniform(curandGenerator, devY, N);
	hiprandDestroyGenerator(curandGenerator); 

	
	CUDA_CHECK_ERROR(hipMemcpy(X, devX, N * sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK_ERROR(hipMemcpy(Y, devY, N * sizeof(float), hipMemcpyDeviceToHost));

	clock_t  start_time = clock();
    float cpu_result = calculationPiCPU(X, Y);
	clock_t  end_time = clock();
	std::cout << "Время на CPU = " << (double)((end_time - start_time) * 1000 / CLOCKS_PER_SEC) << " мсек" << std::endl;
	std::cout << "result: " << cpu_result << std::endl;
	
	int *dev_blocks_counts = 0, *blocks_counts = 0;
	float gpuTime = 0;

	hipEvent_t start;
	hipEvent_t stop;

	int blockDim = 512; 
	int gridDim = N / (128 * blockDim); 


	blocks_counts = (int *)calloc(gridDim, sizeof(int));

	CUDA_CHECK_ERROR(hipMalloc((void **)&dev_blocks_counts, 512 * sizeof(int)));

	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));

	hipEventRecord(start, 0);

	calculationPiGPU << <gridDim, blockDim >> >(devX, devY, dev_blocks_counts);

	
	CUDA_CHECK_ERROR(hipMemcpy(blocks_counts, dev_blocks_counts, gridDim * sizeof(int), hipMemcpyDeviceToHost));

	int countPointsInCircle = 0;
	for (int i = 0; i < gridDim; i++) {
		countPointsInCircle += blocks_counts[i];
	}

	
	float gpu_result = (float) countPointsInCircle * 4 / N;

	
	hipEventRecord(stop, 0);

	
	hipEventSynchronize(stop);

	
	hipEventElapsedTime(&gpuTime, start, stop);

	std::cout << "Время на GPU = " << gpuTime << " мсек" << std::endl;
	std::cout << "result: " << gpu_result << std::endl;

	
	CUDA_CHECK_ERROR(hipEventDestroy(start));
	CUDA_CHECK_ERROR(hipEventDestroy(stop));

	CUDA_CHECK_ERROR(hipFree(devX));
	CUDA_CHECK_ERROR(hipFree(devY));
	CUDA_CHECK_ERROR(hipFree(dev_blocks_counts));

	
	delete X;
	delete Y;

	system("pause");
	return 0;
}
